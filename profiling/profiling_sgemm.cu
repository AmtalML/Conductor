#include "hip/hip_runtime.h"
#include "../Conductor/include/utils.cuh"



namespace Amtal {


typedef __hip_bfloat16 bf16;


__global__ void warmupKernel() {
    __shared__ float s[100];
    s[0] += s[1];
}

}




int main() {

    // Profiling SGEMM Kernel Flops 

    int N = 1 >> 10;
    
    bf16 *host_a = nullptr, *host_b = nullptr, *host_c = nullptr, *copy_host_c = nullptr;

    bf16 *device_a = nullptr, *device_b = nullptr, *device_c = nullptr, *copy_device_c = nullptr;

    host_a = (bf16 *)malloc(sizeof(bf16) * N * N), host_b = (bf16 *)malloc(sizeof(bf16) * N * N);

    host_c = (bf16 *)malloc(sizeof(bf16) * N * N), copy_host_c = (bf16 *)malloc(sizeof(bf16) * N * N);


    CHECK_CUDA(hipMalloc((void **)&device_a, sizeof(bf16) * N * N));
    CHECK_CUDA(hipMalloc((void **)&device_b, sizeof(bf16) * N * N));
    CHECK_CUDA(hipMalloc((void **)&device_c, sizeof(bf16) * N * N));
    CHECK_CUDA(hipMalloc((void **)&copy_device_c, sizeof(bf16) * N * N));

    CHECK_CUDA(hipMemcpy(device_a, host_a, sizeof(bf16) * N * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_b, host_b, sizeof(bf16) * N * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_c, host_c, sizeof(bf16) * N * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(copy_device_c, copy_host_c, sizeof(bf16) * N * N, hipMemcpyHostToDevice));
    
    // warmup kernel
    warmupKernel<<<1024, 1024>>>();

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsed_time;

    int trial_runs = 20;


    // Run kernel 20 times, ret avg of all runs
    for (int i = 0; i < trial_runs;; ++i) {

        hipEventRecord(start, 0);


    

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);

        elapsed_time /= 1000.;


    }
    
    

    hipEventDestroy(&start);
    hipEventDestroy(&stop);

    return 0;

}